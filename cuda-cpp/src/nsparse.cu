#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <CSR.hpp>
#include <nsparse.hpp>

#define LINE_LENGTH_MAX 256

// void release_cpu_amb(sfAMB mat)
// {
//     free(mat.cs);
//     free(mat.cl);
//     free(mat.sellcs_val);
//     free(mat.sellcs_col);
//     free(mat.s_write_permutation);
//     free(mat.s_write_permutation_offset);
// }

// void release_amb(sfAMB mat)
// {
//     hipFree(mat.d_cs);
//     hipFree(mat.d_cl);
//     hipFree(mat.d_sellcs_val);
//     hipFree(mat.d_sellcs_col);
//     hipFree(mat.d_write_permutation);
//     hipFree(mat.d_s_write_permutation);
//     hipFree(mat.d_s_write_permutation_offset);
// }

